#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Eric Brandt, Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include "grayscale.cuh"
#include <iostream>

namespace chrono {
namespace sensor {

__global__ void lidar_clip_kernel(float* buf, int w, int h, float threshold, float default_dist) {
    int out_index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    if (out_index < w * h) {
        // data is packed range,intensity
        if (buf[2 * out_index + 1] < threshold) {
            buf[2 * out_index + 1] = 0;
            buf[2 * out_index] = default_dist;
        }
    }
}

void cuda_lidar_clip(float* buf, int width, int height, float threshold, float default_dist) {
    const int nThreads = 512;
    int nBlocks = (width * height + nThreads - 1) / nThreads;
    lidar_clip_kernel<<<nBlocks, nThreads>>>(buf, width, height, threshold, default_dist);
}

}  // namespace sensor
}  // namespace chrono
