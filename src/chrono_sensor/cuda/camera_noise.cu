#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Eric Brandt, Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "curand_utils.cuh"
#include "camera_noise.cuh"
#include "chrono_sensor/utils/CudaMallocHelper.h"

#include <chrono>
#include <memory>

namespace chrono {
namespace sensor {

// Add random normal noise to the image with constant mean and stdev
__global__ void const_normal_noise_kernel(unsigned char* bufPtr,
                                          int w,
                                          int h,
                                          float mean,
                                          float stdev,
                                          hiprandState_t* rng_states) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < w * h) {
        // hiprand_normal(&rng_states[index]);
        float r_rand = hiprand_normal(&rng_states[index]) * stdev + mean;
        float g_rand = hiprand_normal(&rng_states[index]) * stdev + mean;
        float b_rand = hiprand_normal(&rng_states[index]) * stdev + mean;

        // get pixel values
        unsigned char pix_r = bufPtr[index * 4];
        unsigned char pix_g = bufPtr[index * 4 + 1];
        unsigned char pix_b = bufPtr[index * 4 + 2];

        // convert to float and add noise
        float r = ((float)(pix_r)) / 255.0 + r_rand;
        float g = ((float)(pix_g)) / 255.0 + g_rand;
        float b = ((float)(pix_b)) / 255.0 + b_rand;

        // convert back to char and save in image
        bufPtr[index * 4] = (unsigned char)(r * 255.999);
        bufPtr[index * 4 + 1] = (unsigned char)(g * 255.999);
        bufPtr[index * 4 + 2] = (unsigned char)(b * 255.999);
    }
}

__global__ void pix_dep_noise_kernel(unsigned char* bufPtr,
                                     int w,
                                     int h,
                                     float gain,
                                     float sigma_shot,
                                     float sigma_adc,
                                     hiprandState_t* rng_states) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < w * h) {
        // get pixel values
        unsigned char pix_r = bufPtr[index * 4];
        unsigned char pix_g = bufPtr[index * 4 + 1];
        unsigned char pix_b = bufPtr[index * 4 + 2];

        float r = ((float)(pix_r)) / 255.0;
        float g = ((float)(pix_g)) / 255.0;
        float b = ((float)(pix_b)) / 255.0;

        // hiprand_normal(&rng_states[index]);
        float stdev_r = sqrtf((r * sigma_shot * sigma_shot) + (sigma_adc * sigma_adc));
        float stdev_g = sqrtf((g * sigma_shot * sigma_shot) + (sigma_adc * sigma_adc));
        float stdev_b = sqrtf((b * sigma_shot * sigma_shot) + (sigma_adc * sigma_adc));
        float r_rand = hiprand_normal(&rng_states[index]) * stdev_r;
        float g_rand = hiprand_normal(&rng_states[index]) * stdev_g;
        float b_rand = hiprand_normal(&rng_states[index]) * stdev_b;

        // convert to float and add noise
        r = r + r_rand;
        g = g + g_rand;
        b = b + b_rand;

        // convert back to char and save in image
        bufPtr[index * 4] = (unsigned char)(r * 255.999);
        bufPtr[index * 4 + 1] = (unsigned char)(g * 255.999);
        bufPtr[index * 4 + 2] = (unsigned char)(b * 255.999);
    }
}

void cuda_camera_noise_const_normal(unsigned char* bufPtr,
                                    int width,
                                    int height,
                                    float mean,
                                    float stdev,
                                    hiprandState_t* rng) {
    const int nThreads = 512;
    int nBlocks = (width * height + nThreads - 1) / nThreads;

    const_normal_noise_kernel<<<nBlocks, nThreads>>>(bufPtr, width, height, mean, stdev, rng);
}

void cuda_camera_noise_pixel_dependent(unsigned char* bufPtr,
                                       int width,
                                       int height,
                                       float gain,
                                       float sigma_read,
                                       float sigma_adc,
                                       hiprandState_t* rng) {
    const int nThreads = 512;
    int nBlocks = (width * height + nThreads - 1) / nThreads;

    pix_dep_noise_kernel<<<nBlocks, nThreads>>>(bufPtr, width, height, gain, sigma_read, sigma_adc, rng);
}

}  // namespace sensor
}  // namespace chrono
