#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Eric Brandt, Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include "grayscale.cuh"
#include <iostream>

namespace chrono {
namespace sensor {

// Converts 32bpp ARGB imgIn pixels to 8bpp Grayscale imgOut pixels
__global__ void mean_reduce_kernel(float* bufIn, float* bufOut, int w, int h, int r) {
    int out_index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    int out_hIndex = out_index % w;
    int out_vIndex = out_index / w;

    int d = r * 2 - 1;

    if (out_index < w * h) {
        // reset buffer to zeros
        bufOut[2 * out_index] = 0;
        bufOut[2 * out_index + 1] = 0;

        float sum_range = 0.f;
        float sum_intensity = 0.f;
        int n_contributing = 0;
        // gather up all of our values, take mean and push to output buffer
        for (int i = 0; i < d; i++) {
            for (int j = 0; j < d; j++) {
                int in_index = (d * out_vIndex + i) * d * w + (d * out_hIndex + j);
                sum_intensity += bufIn[2 * in_index + 1];
                if (bufIn[2 * in_index + 1] > 1e-6) {
                    sum_range += bufIn[2 * in_index];
                    n_contributing++;
                }
            }
        }
        if (n_contributing > 0) {
            bufOut[2 * out_index] = sum_range / (n_contributing);
            bufOut[2 * out_index + 1] = sum_intensity / (d * d);
        }
    }
}

// Converts 32bpp ARGB imgIn pixels to 8bpp Grayscale imgOut pixels
__global__ void strong_reduce_kernel(float* bufIn, float* bufOut, int w, int h, int r) {
    int out_index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    int out_hIndex = out_index % w;
    int out_vIndex = out_index / w;

    int d = r * 2 - 1;

    // float* raw_range = new float[d * d];
    // float* raw_intensity = new float[d * d];
    // int raw_id = 0;

    // extract the values we will use in our return distribution
    if (out_index < w * h) {
        float strongest = 0;
        float intensity_at_strongest = 0;

        // perform kernel operation to find max intensity
        float kernel_radius = .05;  // 10 cm total kernel width

        for (int i = 0; i < d; i++) {
            for (int j = 0; j < d; j++) {
                int in_index = (d * out_vIndex + i) * d * w + (d * out_hIndex + j);
                // float range = bufIn[2 * in_index];
                // float intensity = bufIn[2 * in_index + 1];

                float local_range = bufIn[2 * in_index];
                float local_intensity = bufIn[2 * in_index + 1];

                for (int k = 0; k < d; k++) {
                    for (int l = 0; l < d; l++) {
                        int inner_in_index = (d * out_vIndex + k) * d * w + (d * out_hIndex + l);
                        float range = bufIn[2 * inner_in_index];
                        float intensity = bufIn[2 * inner_in_index + 1];

                        if (inner_in_index != in_index && abs(range - local_range) < kernel_radius) {
                            float weight = (kernel_radius - abs(range - local_range)) / kernel_radius;
                            local_intensity += weight * intensity;
                            // norm_val += weight;
                        }
                    }
                }

                local_intensity = local_intensity / (d * d);  // calculating portion of beam here
                if (local_intensity > intensity_at_strongest) {
                    intensity_at_strongest = local_intensity;
                    strongest = local_range;
                }

                // raw_range[raw_id] = bufIn[2 * in_index];
                // raw_intensity[raw_id] = bufIn[2 * in_index + 1];

                // if (raw_id > d * d)
                //     printf("OH NO!\n");
                // raw_id++;
            }
        }

        bufOut[2 * out_index] = strongest;
        bufOut[2 * out_index + 1] = intensity_at_strongest;
    }

    // // essentially performing a linear blur to find range of max intensity
    // for (int i = 0; i < d * d; i++) {
    //     float norm_val = 1;
    //     float local_intensity = raw_intensity[i];
    //     for (int j = 0; j < d * d; j++) {
    //         if (j != i && abs(raw_range[j] - raw_range[i]) < kernel_radius) {
    //             float weight = (kernel_radius - abs(raw_range[j] - raw_range[i])) / kernel_radius;
    //             local_intensity += weight * raw_intensity[j];
    //             norm_val += weight;
    //         }
    //     }
    //     local_intensity = local_intensity / (d * d);  // calculating portion of beam here
    //     if (local_intensity > intensity_at_strongest) {
    //         intensity_at_strongest = local_intensity;
    //         strongest = raw_range[i];
    //     }
    // }
    //
    // // push strongest return
    // bufOut[2 * out_index] = strongest;
    // bufOut[2 * out_index + 1] = intensity_at_strongest;
    // //
    // delete[] raw_range;
    // delete[] raw_intensity;
}

void cuda_lidar_mean_reduce(void* bufIn, void* bufOut, int width, int height, int radius) {
    int w = width / (radius * 2 - 1);
    int h = height / (radius * 2 - 1);
    int numPixels = w * h;
    const int nThreads = 512;
    int nBlocks = (numPixels + nThreads - 1) / nThreads;

    // printf("buffer dimensions: %d,%d\n", w, h);

    // in one shot - each kernel does O(r^2):
    mean_reduce_kernel<<<nBlocks, nThreads>>>((float*)bufIn, (float*)bufOut, w, h, radius);
    // in two shots - each kernel does O(r)
}

void cuda_lidar_strong_reduce(void* bufIn, void* bufOut, int width, int height, int radius) {
    int w = width / (radius * 2 - 1);
    int h = height / (radius * 2 - 1);
    int numPixels = w * h;
    const int nThreads = 512;
    int nBlocks = (numPixels + nThreads - 1) / nThreads;

    // printf("buffer dimensions: %d,%d\n", w, h);

    // in one shot - each kernel does O(r^2):
    strong_reduce_kernel<<<nBlocks, nThreads>>>((float*)bufIn, (float*)bufOut, w, h, radius);
    // in two shots - each kernel does O(r)
}

}  // namespace sensor
}  // namespace chrono
